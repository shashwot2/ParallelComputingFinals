#include <hip/hip_runtime.h>

__global__ void matrixMultiplyNaive(int *A, int *B, int *C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < n && col < n)
    {
        for (int k = 0; k < n; k++)
        {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

extern "C" void performMatrixMultiplication(int *h_A, int *h_B, int *h_C, int N, int threadsPerBlock)
{
    int *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(int);
    int rows_per_process = N / size;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N + threads.x - 1) / threads.x, (rows_per_process + threads.y - 1) / threads.y);


    matrixMultiplyNaive<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipGetLastError();
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}