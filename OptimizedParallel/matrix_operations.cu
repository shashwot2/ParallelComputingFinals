#include "hip/hip_runtime.h"
#include "matrix_operations.h"
#include <hip/hip_runtime.h>

__global__ void matrixMultiplyKernel(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < n && col < n) {
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

void performMatrixMultiplicationCUDA(int *h_A, int *h_B, int *h_C, int n) {
    int *d_A, *d_B, *d_C;
    size_t size = n * n * sizeof(int);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + 15) / 16, (n + 15) / 16);
    matrixMultiplyKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
