#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixMultiplyNaive(int *a, int* b, int *c, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        int sum = 0;
        for (int k = 0; k < width; k++) {
            sum += a[row * width + k] * b[k * width + col];
        }
        c[row * width + col] = sum;
    }
}

extern "C" void performMatrixMultiplication(int *h_A, int* h_B, int *h_C, int local_rows, int N) {
    int *d_A, *d_B, *d_C;
    size_t size_A = local_rows * N * sizeof(int);
    size_t size_B = N * N * sizeof(int);
    size_t size_C = local_rows * N * sizeof(int);

    hipError_t err;

    err = hipMalloc((void **)&d_A, size_A);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc A Error: %s\n", hipGetErrorString(err));
        return;
    }

    err = hipMalloc((void **)&d_B, size_B);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc B Error: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        return;
    }

    err = hipMalloc((void **)&d_C, size_C);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc C Error: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        return;
    }

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (local_rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplyNaive<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Kernel Error: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}