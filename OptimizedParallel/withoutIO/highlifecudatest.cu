#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplyNaive(int *a, int *b, int *c, int width) {
    int k, sum = 0;
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    if (col < width && row < width) {
        for (k = 0; k < width; k++) {
            sum += a[row * width + k] * b[k * width + col];
        }
        c[row * width + col] = sum;
    }
}

extern "C" void performMatrixMultiplication(int N) {
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(int);

    h_A = (int *)malloc(size);
    h_B = (int *)malloc(size);
    h_C = (int *)malloc(size);

    int initA[16] = {1, 2, 3, 4, 1, 2, 3, 4, 1, 2, 3, 4, 1, 2, 3, 4};
    int initB[16] = {1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3, 4, 4, 4, 4};
    memcpy(h_A, initA, size);
    memcpy(h_B, initB, size);

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiplyNaive<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }

    // Synchronize device
    hipDeviceSynchronize();

    // Copy the matrix back to the host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the resulting matrix C
    printf("Resulting matrix C:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", h_C[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
}

int main() {
    int N = 4;  // Dimension of the matrix
    performMatrixMultiplication(N);
    return 0;
}
