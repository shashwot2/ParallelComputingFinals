#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixMulKernel(int *a, int *b, int *c, int width, int local_width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < local_width && col < width) {
        int sum = 0;
        for (int k = 0; k < width; k++) {
            sum += a[row * width + k] * b[k * width + col];
        }
        c[row * width + col] = sum;
    }
}

extern "C" void matrixMulCUDA(int *h_a, int *h_b, int *h_c, int width, int local_width) {
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, local_width * width * sizeof(int));
    hipMalloc((void **)&d_b, width * width * sizeof(int));
    hipMalloc((void **)&d_c, local_width * width * sizeof(int));

    hipMemcpy(d_a, h_a, local_width * width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, width * width * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (local_width + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, width, local_width);

    hipMemcpy(h_c, d_c, local_width * width * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}