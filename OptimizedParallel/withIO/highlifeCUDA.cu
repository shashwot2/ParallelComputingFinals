#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixMulKernel(int *a, int* b, int *c, int width, int local_width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < local_width && col < width) {
        int sum = 0;
        for (int k = 0; k < width; k++) {
            sum += a[row * width + k] * b[k * width + col];
        }
        c[row * width + col] = sum;
    }
}

extern "C" void matrixMulCUDA(int *h_a, int* h_b, int *h_c, int width, int local_width) {
    int *d_a, *d_b, *d_c;
    hipError_t err;

    err = hipMalloc((void **)&d_a, local_width * width * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_b, width * width * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_c, local_width * width * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory for matrix C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_a, h_a, local_width * width * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy matrix A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_b, h_b, width * width * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy matrix B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (local_width + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, width, local_width);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch matrix multiplication kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_c, d_c, local_width * width * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy matrix C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}